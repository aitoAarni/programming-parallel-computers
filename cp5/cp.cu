#include <vector>
#include <iostream>
#include <cmath>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>

static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)
float zeroNormalized[160000000];
float squareNormalized[160000000];
float transpose[160000000];
/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/

__global__ void mykernel(int ny, int nx, const float *data, const float *transpose, float *result) {
    int bx = blockIdx.x * blockDim.x;
    int by = blockIdx.y * blockDim.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    if (bx + tx >= ny || by + ty >= ny || by + ty > bx + tx) return;
    float v1[8];
    float v2[8];
    float vv[8][8];
    for (int y = 0; y < 8; y++) {
        for (int x = 0; x < 8; x++) {
            vv[y][x] = 0;
        }
    }
    for (int k = 0; k < nx; k++) {
        for (int i = 0; i < 8; i++) {
            int v1Col = by + ty + i * 8;
            int v2Col = bx + tx + i * 8;
            if (v1Col >= ny || v2Col >= ny) break;
            v1[i] = transpose[v1Col + k * ny];
            v2[i] = transpose[v2Col + k * ny];

        }
        for (int y = 0; y < 8; y++) {
            for (int x = 0; x < 8; x++) {
                vv[y][x] += v1[y] * v2[x];
            }
        }
    }
    for (int y = 0; y < 8; y++) {
        int j = by + ty + y * 8;
        if (j >= ny) return;
        for (int x = 0; x < 8; x++) {
            int i = bx + tx + x * 8; 
            if (i >= ny) break;
            result[j * ny + i] = vv[y][x];
        }
    }
}

static inline int divup(int a, int b) {
    return (a + b - 1)/b;
}

static inline int roundup(int a, int b) {
    return divup(a, b) * b;
}

void correlate(int ny, int nx, const float *data, float *result) {
    std::vector<float> means(ny, 0.0);
    std::vector<float> squareSums(ny, 0.0);
    for (int y = 0; y < ny; y++) {
        float sum = 0;
        for (int x = 0; x < nx; x++) {
            sum += data[x+y*nx];
        }
        means[y] = sum / nx;
    }
    
    for (int y = 0; y < ny; y++) {
        for (int x = 0; x < nx; x++) {
           zeroNormalized[x+y*nx] = data[x+y*nx] - means[y]; 
        }
    }
    for (int y = 0; y < ny; y++) {
        for (int x = 0; x < nx; x++) {
            squareSums[y] += std::pow(zeroNormalized[x+y*nx], 2);           
        }
    }
    for (int y = 0; y < ny; y++) {
        for (int x = 0; x < nx; x++) {
            squareNormalized[x + y * nx] = zeroNormalized[x+y*nx] / std::sqrt(squareSums[y]);
            transpose[x * ny + y] = squareNormalized[x + y * nx];
        }
    }

    float* dGPU = NULL;
    float* tGPU = NULL;
    CHECK(hipMalloc((void**)&dGPU, nx * ny * sizeof(float)));
    CHECK(hipMalloc((void**)&tGPU, nx * ny * sizeof(float)));
    float* rGPU = NULL;
    CHECK(hipMalloc((void**)&rGPU, ny * ny * sizeof(float)));
    CHECK(hipMemset(rGPU, 0, ny * ny * sizeof(float)));
    CHECK(hipMemcpy(dGPU, squareNormalized, nx * ny * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(tGPU, transpose, nx * ny * sizeof(float), hipMemcpyHostToDevice));

    dim3 dimBlock(8, 8);
    dim3 dimGrid(divup(ny, 64), divup(ny, 64));
    mykernel<<<dimGrid, dimBlock>>>(ny, nx, dGPU, tGPU ,rGPU);
    CHECK(hipGetLastError());

    CHECK(hipMemcpy(result, rGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dGPU));
    CHECK(hipFree(tGPU));
    CHECK(hipFree(rGPU));

}
