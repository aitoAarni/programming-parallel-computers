#include <vector>
#include <iostream>
#include <cmath>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>

using namespace std::chrono;
static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)
float d[160000000];
float transpose[160000000];
/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/

__global__ void mykernel(int nn, int ny, int nx, const float *transpose, float *result) {
    int bx = blockIdx.x * 64;
    int by = blockIdx.y * 64;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    if (bx + tx >= ny || by + ty >= ny || by > bx) return;
    float v1[8];
    float v2[8];
    float vv[8][8];
    for (int y = 0; y < 8; y++) {
        for (int x = 0; x < 8; x++) {
            vv[y][x] = 0;
        }
    }
    for (int k = 0; k < nx; k++) {
        for (int i = 0; i < 8; i++) {
            int v1Col = by + ty + i * 8;
            int v2Col = bx + tx + i * 8;
            v1[i] = transpose[v1Col + k * nn];
            v2[i] = transpose[v2Col + k * nn];

        }
        for (int y = 0; y < 8; y++) {
            for (int x = 0; x < 8; x++) {
                vv[y][x] += v1[y] * v2[x];
            }
        }
    }
    for (int y = 0; y < 8; y++) {
        int j = by + ty + y * 8;
        if (j >= ny) return;
        for (int x = 0; x < 8; x++) {
            int i = bx + tx + x * 8; 
            if (i >= ny) break;
            result[j * ny + i] = vv[y][x];
        }
    }
}

static inline int divup(int a, int b) {
    return (a + b - 1)/b;
}

static inline int roundup(int a, int b) {
    return divup(a, b) * b;
}

__global__ void preprocess(int ny, int nx, int nn, const float* data, float* d, float* transpose) {
    int y = blockIdx.x;    
    int tx = threadIdx.x;  

    __shared__ float sum_shared;
    __shared__ float sqsum_shared;

    float partial_sum = 0.0f;
    float partial_sqsum = 0.0f;

    for (int x = tx; x < nx; x += blockDim.x) {
        partial_sum += data[x + y * nx];
    }

    __shared__ float temp[256]; 
    temp[tx] = partial_sum;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tx < stride) temp[tx] += temp[tx + stride];
        __syncthreads();
    }

    if (tx == 0) sum_shared = temp[0];
    __syncthreads();

    float mean = sum_shared / nx;

    // Step 2: subtract mean and compute squared sum
    for (int x = tx; x < nx; x += blockDim.x) {
        float v = data[x + y * nx] - mean;
        d[x + y * nx] = v;
        partial_sqsum += v * v;
    }

    temp[tx] = partial_sqsum;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tx < stride) temp[tx] += temp[tx + stride];
        __syncthreads();
    }

    if (tx == 0) sqsum_shared = temp[0];
    __syncthreads();

    float norm = sqrtf(sqsum_shared);

    // Step 3: normalize and store transpose
    for (int x = tx; x < nx; x += blockDim.x) {
        float v = d[x + y * nx] / norm;
        d[x + y * nx] = v;
        transpose[x * nn + y] = v;
    }
}


void correlate(int ny, int nx, const float *data, float *result) {
    auto start1 = high_resolution_clock::now();
    float* dataGPU;
    CHECK(hipMalloc(&dataGPU, nx * ny * sizeof(float)));
    CHECK(hipMemcpy(dataGPU, data, nx * ny * sizeof(float), hipMemcpyHostToDevice));

    int nn = roundup(ny, 64);
    float* dGPU;
    float* tGPU;
    CHECK(hipMalloc(&dGPU, nx * ny * sizeof(float)));
    CHECK(hipMalloc(&tGPU, nx * nn * sizeof(float)));

    dim3 preBlock(256);
    dim3 preGrid(ny);  // one block per row
    preprocess<<<preGrid, preBlock>>>(ny, nx, nn, dataGPU, dGPU, tGPU);
    CHECK(hipGetLastError());

    auto end1 = high_resolution_clock::now();
    auto start2 = high_resolution_clock::now();

    float* rGPU = NULL;
    CHECK(hipMalloc((void**)&rGPU, ny * ny * sizeof(float)));
    CHECK(hipMemset(rGPU, 0, ny * ny * sizeof(float)));
    auto end2 = high_resolution_clock::now();

    dim3 dimBlock(8, 8);
    dim3 dimGrid(divup(ny, 64), divup(ny, 64));
    auto start3 = high_resolution_clock::now();

    mykernel<<<dimGrid, dimBlock>>>(nn, ny, nx, tGPU, rGPU);
    auto end3 = high_resolution_clock::now();

    CHECK(hipGetLastError());

    CHECK(hipMemcpy(result, rGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost));
    
    // Free all GPU memory allocations
    CHECK(hipFree(dataGPU));  // Fix: Added missing free for dataGPU
    CHECK(hipFree(dGPU));     // Fix: Added missing free for dGPU
    CHECK(hipFree(tGPU));
    CHECK(hipFree(rGPU));
    
    auto duration1 = std::chrono::duration<double, std::milli>(end1 - start1).count();
    auto duration2 = std::chrono::duration<double, std::milli>(end2 - start2).count();
    auto duration3 = std::chrono::duration<double, std::milli>(end3 - start3).count();

    // std::printf("first: %f  second %f  third: %f", duration1, duration2, duration3);
}